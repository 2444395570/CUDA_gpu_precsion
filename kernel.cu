#include <iostream>
#include <hip/hip_runtime.h>


#include <memory>



/*
如果在应用程序中内核需要与CPU紧密交互，那么你可能希望内核运行在与CPU共享系统内存的集成CPU上。了解设备是否支持
双精度浮点操作并为你的应用程序设置该设备。
使用cudaDeviceProp结构体中的两个属性来帮助识别设备是否支持双精度操作。这两个属性时major和minor。如果
major大于1而minor大于3，那么该设备将支持双精度操作。
因此，程序的device_property结果包含这两个值。CUDA还提供了cudaChooseDevice API,帮助选择具有特定属性的设备。
*/


int main(void) {
	int device;
	hipDeviceProp_t device_property;
	hipGetDevice(&device);
	printf("ID of device:%d\n", device);
	memset(&device_property, 0, sizeof(hipDeviceProp_t));
	device_property.major = 1;
	device_property.minor = 3;
	hipChooseDevice(&device, &device_property);
	printf("ID of device which supports double precision is:%d\n", device);
	hipSetDevice(device);
}