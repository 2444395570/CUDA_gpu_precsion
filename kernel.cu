#include <iostream>
#include <hip/hip_runtime.h>


#include <memory>



/*
�����Ӧ�ó������ں���Ҫ��CPU���ܽ�������ô�����ϣ���ں���������CPU����ϵͳ�ڴ�ļ���CPU�ϡ��˽��豸�Ƿ�֧��
˫���ȸ��������Ϊ���Ӧ�ó������ø��豸��
ʹ��cudaDeviceProp�ṹ���е���������������ʶ���豸�Ƿ�֧��˫���Ȳ���������������ʱmajor��minor�����
major����1��minor����3����ô���豸��֧��˫���Ȳ�����
��ˣ������device_property�������������ֵ��CUDA���ṩ��cudaChooseDevice API,����ѡ������ض����Ե��豸��
*/


int main(void) {
	int device;
	hipDeviceProp_t device_property;
	hipGetDevice(&device);
	printf("ID of device:%d\n", device);
	memset(&device_property, 0, sizeof(hipDeviceProp_t));
	device_property.major = 1;
	device_property.minor = 3;
	hipChooseDevice(&device, &device_property);
	printf("ID of device which supports double precision is:%d\n", device);
	hipSetDevice(device);
}